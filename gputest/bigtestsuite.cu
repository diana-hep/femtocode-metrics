#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>

#define NUM_POINTS 33554432    // 1 GB of 32-bit floats

float cpu_dataset[NUM_POINTS];
float cpu2_dataset[NUM_POINTS];

__global__ void hitAtomic(float* where) {
  atomicAdd(where, 1.0);
}

__global__ void hitAtomicBy32(float* where) {
  atomicAdd(&where[threadIdx.x % 32], 1.0);
}

__global__ void hitAtomicThreadLocal(float* where) {
  atomicAdd(&where[threadIdx.x], 1.0);
}

__global__ void hitNaiveThreadLocal(float* where) {
  where[threadIdx.x] += 1.0;
}

__global__ void inplaceOperation(float* data) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  data[id] = 1.0 - data[id];
}

__global__ void immutableOperation(float* datain, float* dataout) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  dataout[id] = 1.0 - datain[id];
}

__global__ void constImmutableOperation(const float* datain, float* dataout) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  dataout[id] = 1.0 - datain[id];
}

int main(int argc, char** argv) {
  srand(12345);
  for (int i = 0;  i < NUM_POINTS;  i++)
    cpu_dataset[i] = ((float)rand()) / RAND_MAX;

  struct hipDeviceProp_t cdp;
  hipGetDeviceProperties(&cdp, 0);
  std::cout << "Device at 0:" << std::endl;
  std::cout << "    name: " << cdp.name << std::endl;
  std::cout << "    totalGlobalMem: " << cdp.totalGlobalMem / 1024.0 / 1024.0 / 1024.0 << " GB" << std::endl;
  std::cout << "    sharedMemPerBlock: " << cdp.sharedMemPerBlock / 1024.0 << " kB" << std::endl;
  std::cout << "    regsPerBlock: " << cdp.regsPerBlock << std::endl;
  std::cout << "    warpSize: " << cdp.warpSize << std::endl;
  std::cout << "    memPitch: " << cdp.memPitch / 1024.0 / 1024.0 / 1024.0 << " GB" << std::endl;
  std::cout << "    maxThreadsPerBlock: " << cdp.maxThreadsPerBlock << std::endl;
  std::cout << "    maxThreadsDim: " << cdp.maxThreadsDim[0] << " " << cdp.maxThreadsDim[1] << " " << cdp.maxThreadsDim[2] << " " << std::endl;
  std::cout << "    maxGridSize: " << cdp.maxGridSize[0] << " " << cdp.maxGridSize[1] << " " << cdp.maxGridSize[2] << " " << std::endl;
  std::cout << "    totalConstMem: " << cdp.totalConstMem / 1024.0 << " kB" << std::endl;
  std::cout << "    version: " << cdp.major << "." << cdp.minor << std::endl;
  std::cout << "    clockRate: " << cdp.clockRate / 1000.0 << " MHz" << std::endl;
  std::cout << "    textureAlignment: " << cdp.textureAlignment << std::endl;
  std::cout << "    deviceOverlap: " << (cdp.deviceOverlap ? "true" : "false") << std::endl;
  std::cout << "    multiProcessorCount: " << cdp.multiProcessorCount << std::endl;
  std::cout << "    kernelExecTimeoutEnabled: " << (cdp.kernelExecTimeoutEnabled ? "true" : "false") << std::endl;
  std::cout << "    integrated: " << (cdp.integrated ? "true" : "false") << std::endl;
  std::cout << "    canMapHostMemory: " << (cdp.canMapHostMemory ? "true" : "false") << std::endl;
  std::cout << "    computeMode: " << (cdp.computeMode == hipComputeModeDefault ? "hipComputeModeDefault" : (cdp.computeMode == hipComputeModeExclusive ? "hipComputeModeExclusive" : (cdp.computeMode == hipComputeModeProhibited ? "hipComputeModeProhibited" : "unknown"))) << std::endl;
  std::cout << "    concurrentKernels: " << (cdp.concurrentKernels ? "true" : "false") << std::endl;
  std::cout << "    ECCEnabled: " << (cdp.ECCEnabled ? "true" : "false") << std::endl;
  std::cout << "    pciBusID: " << cdp.pciBusID << std::endl;
  std::cout << "    pciDeviceID: " << cdp.pciDeviceID << std::endl;
  std::cout << "    tccDriver: " << (cdp.tccDriver ? "true" : "false") << std::endl;
  std::cout << std::endl;

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      memcpy(cpu2_dataset, cpu_dataset, NUM_POINTS * 4);
    }
    std::cout << "1 GB host -> host: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  float* gpu_dataset;
  float* gpu2_dataset;
  hipMalloc((void**)&gpu_dataset, NUM_POINTS * 4);
  hipMalloc((void**)&gpu2_dataset, NUM_POINTS * 4);

  std::cout << "check " << cpu_dataset[0] << " " << cpu_dataset[1] << " " << cpu_dataset[2] << std::endl;

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hipMemcpy(gpu_dataset, cpu_dataset, NUM_POINTS * 4, hipMemcpyHostToDevice);
      hipDeviceSynchronize();
    }
    std::cout << "1 GB host -> device: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      inplaceOperation<<<NUM_POINTS / cdp.maxThreadsPerBlock, cdp.maxThreadsPerBlock>>>(gpu_dataset);
      hipDeviceSynchronize();
    }
    std::cout << "1 GB device in-place operation: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      immutableOperation<<<NUM_POINTS / cdp.maxThreadsPerBlock, cdp.maxThreadsPerBlock>>>(gpu_dataset, gpu2_dataset);
      hipDeviceSynchronize();
    }
    std::cout << "1 GB device immutable operation: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      constImmutableOperation<<<NUM_POINTS / cdp.maxThreadsPerBlock, cdp.maxThreadsPerBlock>>>(gpu_dataset, gpu2_dataset);
      hipDeviceSynchronize();
    }
    std::cout << "1 GB device const immutable operation: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hipMemcpy(cpu_dataset, gpu_dataset, NUM_POINTS * 4, hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    std::cout << "1 GB device -> host: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  std::cout << "check " << cpu_dataset[0] << " " << cpu_dataset[1] << " " << cpu_dataset[2] << std::endl;

  float* pinned_dataset;
  hipHostMalloc((void**)&pinned_dataset, NUM_POINTS * 4);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      memcpy(pinned_dataset, cpu_dataset, NUM_POINTS * 4);
    }
    std::cout << "1 GB host -> pinned: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  float* mapped_dataset;
  hipHostGetDevicePointer((void**)&mapped_dataset, (void*)pinned_dataset, 0);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      inplaceOperation<<<NUM_POINTS / cdp.maxThreadsPerBlock, cdp.maxThreadsPerBlock>>>(mapped_dataset);
      hipDeviceSynchronize();
    }
    std::cout << "1 GB device in-place operation: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      immutableOperation<<<NUM_POINTS / cdp.maxThreadsPerBlock, cdp.maxThreadsPerBlock>>>(mapped_dataset, gpu2_dataset);
      hipDeviceSynchronize();
    }
    std::cout << "1 GB device immutable operation: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      constImmutableOperation<<<NUM_POINTS / cdp.maxThreadsPerBlock, cdp.maxThreadsPerBlock>>>(mapped_dataset, gpu2_dataset);
      hipDeviceSynchronize();
    }
    std::cout << "1 GB device const immutable operation: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      memcpy(cpu_dataset, pinned_dataset, NUM_POINTS * 4);
    }
    std::cout << "1 GB pinned -> host: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  std::cout << "check " << cpu_dataset[0] << " " << cpu_dataset[1] << " " << cpu_dataset[2] << std::endl;

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hipMemcpy(gpu2_dataset, gpu_dataset, NUM_POINTS * 4, hipMemcpyDeviceToDevice);
      hipDeviceSynchronize();
    }
    std::cout << "1 GB device -> device: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  const float atomic_init[32] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
  float* global_atomics;
  hipMalloc((void**)&global_atomics, 32 * 4);
  float cpu_atomics[32];

  hipMemcpy(global_atomics, atomic_init, 32 * 4, hipMemcpyHostToDevice);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hitAtomic<<<32768, 32>>>(global_atomics);
      hipDeviceSynchronize();
    }
    std::cout << "hit atomics on global <<<32768, 32>>>: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  hipMemcpy(cpu_atomics, global_atomics, 32 * 4, hipMemcpyDeviceToHost);

  std::cout << "check for " << 32768 * 32 * 5 << ": ";
  for (int i = 0;  i < 32;  i++)
    std::cout << cpu_atomics[i] << " ";
  std::cout << std::endl;

  hipMemcpy(global_atomics, atomic_init, 32 * 4, hipMemcpyHostToDevice);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hitAtomic<<<32768, 64>>>(global_atomics);
      hipDeviceSynchronize();
    }
    std::cout << "hit atomics on global <<<32768, 64>>>: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  hipMemcpy(cpu_atomics, global_atomics, 32 * 4, hipMemcpyDeviceToHost);

  std::cout << "check for " << 32768 * 64 * 5 << ": ";
  for (int i = 0;  i < 32;  i++)
    std::cout << cpu_atomics[i] << " ";
  std::cout << std::endl;

  hipMemcpy(global_atomics, atomic_init, 32 * 4, hipMemcpyHostToDevice);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hitAtomic<<<32768, cdp.maxThreadsPerBlock>>>(global_atomics);
      hipDeviceSynchronize();
    }
    std::cout << "hit atomics on global <<<32768, " << cdp.maxThreadsPerBlock << ">>>: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  hipMemcpy(cpu_atomics, global_atomics, 32 * 4, hipMemcpyDeviceToHost);

  std::cout << "check for " << 32768 * cdp.maxThreadsPerBlock * 5 << ": ";
  for (int i = 0;  i < 32;  i++)
    std::cout << cpu_atomics[i] << " ";
  std::cout << std::endl;

  hipMemcpy(global_atomics, atomic_init, 32 * 4, hipMemcpyHostToDevice);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hitAtomicBy32<<<32768, 32>>>(global_atomics);
      hipDeviceSynchronize();
    }
    std::cout << "hit atomics by 32 on global <<<32768, 32>>>: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  hipMemcpy(cpu_atomics, global_atomics, 32 * 4, hipMemcpyDeviceToHost);

  std::cout << "check for " << 32768 * 32 * 5 / 32 << ": ";
  for (int i = 0;  i < 32;  i++)
    std::cout << cpu_atomics[i] << " ";
  std::cout << std::endl;

  hipMemcpy(global_atomics, atomic_init, 32 * 4, hipMemcpyHostToDevice);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hitAtomicBy32<<<32768, 64>>>(global_atomics);
      hipDeviceSynchronize();
    }
    std::cout << "hit atomics by 32 on global <<<32768, 64>>>: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  hipMemcpy(cpu_atomics, global_atomics, 32 * 4, hipMemcpyDeviceToHost);

  std::cout << "check for " << 32768 * 64 * 5 / 32 << ": ";
  for (int i = 0;  i < 32;  i++)
    std::cout << cpu_atomics[i] << " ";
  std::cout << std::endl;

  hipMemcpy(global_atomics, atomic_init, 32 * 4, hipMemcpyHostToDevice);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hitAtomicBy32<<<32768, cdp.maxThreadsPerBlock>>>(global_atomics);
      hipDeviceSynchronize();
    }
    std::cout << "hit atomics by 32 on global <<<32768, " << cdp.maxThreadsPerBlock << ">>>: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  hipMemcpy(cpu_atomics, global_atomics, 32 * 4, hipMemcpyDeviceToHost);

  std::cout << "check for " << 32768 * cdp.maxThreadsPerBlock * 5 / 32 << ": ";
  for (int i = 0;  i < 32;  i++)
    std::cout << cpu_atomics[i] << " ";
  std::cout << std::endl;

  float* atomic_init2 = (float*)malloc(cdp.maxThreadsPerBlock * 4);
  float* global_atomics2;
  hipMalloc((void**)&global_atomics2, cdp.maxThreadsPerBlock * 4);
  float* cpu_atomics2 = new float[cdp.maxThreadsPerBlock];

  for (int i = 0;  i < cdp.maxThreadsPerBlock;  i++)
    atomic_init2[i] = 0.0;

  hipMemcpy(global_atomics2, atomic_init2, cdp.maxThreadsPerBlock * 4, hipMemcpyHostToDevice);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hitNaiveThreadLocal<<<32768, cdp.maxThreadsPerBlock>>>(global_atomics2);
      hipDeviceSynchronize();
    }
    std::cout << "hit naive thread local by " << cdp.maxThreadsPerBlock << " on global <<<32768, " << cdp.maxThreadsPerBlock << ">>>: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  hipMemcpy(cpu_atomics2, global_atomics2, cdp.maxThreadsPerBlock * 4, hipMemcpyDeviceToHost);

  std::cout << "check for " << 32768 * 5 << ": ";
  for (int i = 0;  i < cdp.maxThreadsPerBlock;  i++)
    std::cout << cpu_atomics2[i] << " ";
  std::cout << std::endl;

  for (int i = 0;  i < cdp.maxThreadsPerBlock;  i++)
    atomic_init2[i] = 0.0;

  hipMemcpy(global_atomics2, atomic_init2, cdp.maxThreadsPerBlock * 4, hipMemcpyHostToDevice);

  for (int i = 0;  i < 5;  i++) {
    std::clock_t startTime = std::clock();
    for (int j = 0;  j < 1000;  j++) {
      hitAtomicThreadLocal<<<32768, cdp.maxThreadsPerBlock>>>(global_atomics2);
      hipDeviceSynchronize();
    }
    std::cout << "hit atomic thread local by " << cdp.maxThreadsPerBlock << " on global <<<32768, " << cdp.maxThreadsPerBlock << ">>>: " << (std::clock() - startTime) / (double) CLOCKS_PER_SEC << " ms" << std::endl;
  }

  hipMemcpy(cpu_atomics2, global_atomics2, cdp.maxThreadsPerBlock * 4, hipMemcpyDeviceToHost);

  std::cout << "check for " << 32768 * 5 << ": ";
  for (int i = 0;  i < cdp.maxThreadsPerBlock;  i++)
    std::cout << cpu_atomics2[i] << " ";
  std::cout << std::endl;

  return 0;
}
